#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include <unordered_map>
#include <float.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/transform_scan.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include "SCRIMP.h"
#include "tile.h"

using std::vector;
using std::unordered_map;
using std::make_pair;

namespace SCRIMP {


__global__ void cross_correlation_to_ed(float *profile, unsigned int n, unsigned int m) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) {
        profile[tid] = sqrt(max(2*(1 - profile[tid]), 0.0)) * sqrt((double)m);
    }
}

//This kernel computes a sliding mean with specified window size and a corresponding prefix sum array (A)
template<class DTYPE>
__global__ void sliding_mean(DTYPE* pref_sum,  size_t window, size_t size, DTYPE* means)
{
    const DTYPE coeff = 1.0 / (DTYPE) window;
    size_t a = blockIdx.x * blockDim.x + threadIdx.x;
    size_t b = blockIdx.x * blockDim.x + threadIdx.x + window;

    if(a == 0){
        means[a] = pref_sum[window - 1] * coeff;
    }
    if(a < size - 1){
        means[a + 1] = (pref_sum[b] - pref_sum[a]) * coeff;
    }
}

template<class DTYPE>
__global__ void sliding_norm(DTYPE* cumsumsqr, unsigned int window, unsigned int size, DTYPE* norms) {
    const DTYPE coeff = 1 / (DTYPE) window;
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.x * blockDim.x + threadIdx.x + window;
    if (a == 0) {
        norms[a] = 1 / sqrt(cumsumsqr[window - 1]);
    }
    else if (b < size + window) {
        norms[a] = 1 / sqrt(cumsumsqr[b - 1] - cumsumsqr[a - 1]);
    }
}

template<class DTYPE>
__global__ void sliding_dfdg(const DTYPE *T, const DTYPE *means, DTYPE *df, DTYPE *dg, const int m, const int n) {
    const DTYPE half = 1.0 / (DTYPE) 2.0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n - 1) {
        df[tid] = (T[tid + m] - T[tid]) * half;
        dg[tid] = (T[tid + m] - means[tid + 1]) + (T[tid] - means[tid]);
    }
}

__global__ void __launch_bounds__(512,4) 
fastinvnorm(double *norm, const double *mean, const double *T, int m, int n) {
   
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int jump = ceil(n / (double) (blockDim.x * gridDim.x));
    int start = jump * tid;
    int end = jump * (tid + 1);
    end = min(end,n);
    if(start >= n) {
        return;
    }
    double sum = 0;
    for(int i = 0; i < m; ++i){ 
        double val = T[i + start] - mean[start];
        sum += val * val;
    }
    norm[start] = sum;
    
    for(int i = start+1; i < end; ++i) {
            norm[i] = norm[i - 1]  + ((T[i-1] - mean[i-1]) + (T[i + m - 1] - mean[i])) * (T[i + m - 1] - T[i - 1]);
    }
    for(int i = start; i < end; ++i) {
        norm[i] = 1.0 / sqrt(norm[i]);
    }
}


void compute_statistics(const double *T, double *norms, double *df, double *dg,
                        double *means, size_t n, size_t m, hipStream_t s, double *scratch)
{
    square<double> sqr;
    dim3 grid(ceil(n / (double) 512), 1,1);
    dim3 block(512, 1, 1);
    
    gpuErrchk(hipPeekAtLastError());
    
    thrust::device_ptr<const double> dev_ptr_T = thrust::device_pointer_cast(T);
    thrust::device_ptr<double> dev_ptr_scratch = thrust::device_pointer_cast(scratch);

    // Compute prefix sum in scratch
    thrust::inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T, dev_ptr_T + n + m - 1,
                           dev_ptr_scratch, thrust::plus<double>());
    gpuErrchk(hipPeekAtLastError());
    // Use prefix sum to compute sliding mean
    sliding_mean<double><<<grid, block, 0, s>>>(scratch, m, n, means);
    gpuErrchk(hipPeekAtLastError());
    
    // Compute differential values
    sliding_dfdg<double><<<grid, block, 0, s>>>(T, means, df,dg,m,n);
    gpuErrchk(hipPeekAtLastError());
    
    // Compute prefix sum of squares in scratch
    thrust::transform_inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T, dev_ptr_T + n + m - 1,
                                     dev_ptr_scratch, sqr,thrust::plus<double>());
    gpuErrchk(hipPeekAtLastError());
    
    // This will be kind of slow on the GPU, may cause latency between tiles
    int workers = n / m + 1;
    fastinvnorm<<<dim3(ceil(workers / (double)512),1,1), dim3(512,1,1), 0, s>>>(norms, means, T, m, n);
    gpuErrchk(hipPeekAtLastError());
    
}

SCRIMPError_t SCRIMP_Operation::init()
{
    for (auto device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        
        T_A_dev.insert(make_pair(device, (double*) 0));
        T_B_dev.insert(make_pair(device, (double*) 0));
        QT_dev.insert(make_pair(device, (double*) 0));
        means_A.insert(make_pair(device, (double*) 0));
        means_B.insert(make_pair(device, (double*) 0));
        norms_A.insert(make_pair(device, (double*) 0));
        norms_B.insert(make_pair(device, (double*) 0));
        df_A.insert(make_pair(device, (double*) 0));
        df_B.insert(make_pair(device, (double*) 0));
        dg_A.insert(make_pair(device, (double*) 0));
        dg_B.insert(make_pair(device, (double*) 0));
        profile_A_dev.insert(make_pair(device,(float*) NULL));
        profile_B_dev.insert(make_pair(device,(float*) NULL));
        profile_A_merged.insert(make_pair(device,(unsigned long long int*) NULL));
        profile_B_merged.insert(make_pair(device,(unsigned long long int*) NULL));
        profile_idx_A_dev.insert(make_pair(device,(unsigned int *) NULL));
        profile_idx_B_dev.insert(make_pair(device,(unsigned int *) NULL));
        scratchpad.insert(make_pair(device, (double*) NULL));

        hipMalloc(&T_A_dev.at(device), sizeof(double) * tile_size);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&T_B_dev.at(device), sizeof(double) * tile_size);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_A_dev.at(device), sizeof(float) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_B_dev.at(device), sizeof(float) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_idx_A_dev.at(device), sizeof(unsigned int) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_idx_B_dev.at(device), sizeof(unsigned int) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&QT_dev.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&means_A.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&means_B.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&norms_A.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&norms_B.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&df_A.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&df_B.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&dg_A.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&dg_B.at(device), sizeof(double) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_A_merged.at(device), sizeof(unsigned long long int) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&profile_B_merged.at(device), sizeof(unsigned long long int) * tile_n);
        gpuErrchk(hipPeekAtLastError());
        hipMalloc(&scratchpad.at(device), sizeof(double) * tile_size);
        scratch[device] = new fft_precompute_helper(tile_size, m, true);
        hipEvent_t st, ed, copy;
        hipEventCreate(&ed);
        gpuErrchk(hipPeekAtLastError());
        hipEventCreate(&st);
        gpuErrchk(hipPeekAtLastError());
        hipEventCreate(&copy);
        gpuErrchk(hipPeekAtLastError());
        
        clocks_start.emplace(device, st);
        clocks_end.emplace(device, ed);
        copy_to_host_done.emplace(device, copy);
        hipStream_t s;
        hipStreamCreate(&s);
        gpuErrchk(hipPeekAtLastError());
        streams.emplace(device, s);
    }
    return SCRIMP_NO_ERROR;

}

SCRIMPError_t SCRIMP_Operation::destroy()
{
    for (auto device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipFree(T_A_dev[device]);
        hipFree(T_B_dev[device]);
        hipFree(QT_dev[device]);
        hipFree(means_A[device]);
        hipFree(means_B[device]);
        hipFree(norms_A[device]);
        hipFree(norms_B[device]);
        hipFree(df_A[device]);
        hipFree(df_B[device]);
        hipFree(dg_A[device]);
        hipFree(dg_B[device]);
        hipFree(profile_A_dev[device]);
        hipFree(profile_B_dev[device]);
        hipFree(profile_A_merged[device]);
        hipFree(profile_B_merged[device]);
        hipFree(profile_idx_A_dev[device]);
        hipFree(profile_idx_B_dev[device]);
        hipFree(scratchpad.at(device));
        delete scratch[device];
        hipEventDestroy(clocks_start[device]);
        hipEventDestroy(clocks_end[device]);
        hipEventDestroy(copy_to_host_done[device]);
        hipStreamDestroy(streams.at(device));
    }
    return SCRIMP_NO_ERROR;

}

SCRIMPError_t SCRIMP_Operation::do_tile(SCRIMPTileType t, size_t t_size_x, size_t t_size_y, size_t start_x, size_t start_y, int device, const vector<double> &T_h, const vector<float> &profile_h, const vector<unsigned int> &profile_idx_h)
{ 
        MPIDXCombine combiner;
        SCRIMPError_t err;
        size_t t_n_x = t_size_x - m + 1;
        size_t t_n_y = t_size_y - m + 1;
        printf("tile type = %d start_pos = [%lu, %lu]...\n", t, start_y, start_x);
        hipMemcpyAsync(T_A_dev[device], T_h.data() + start_x, sizeof(double) * t_size_x, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(T_B_dev[device], T_h.data() + start_y, sizeof(double) * t_size_y, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_A_dev[device], profile_h.data() + start_x, sizeof(float) * t_n_x, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_B_dev[device], profile_h.data() + start_y, sizeof(float) * t_n_y, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_idx_A_dev[device], profile_idx_h.data() + start_x, sizeof(unsigned int) * t_n_x, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profile_idx_B_dev[device], profile_idx_h.data() + start_y, sizeof(unsigned int) * t_n_y, hipMemcpyHostToDevice, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        
        // FIXME?: Computing the sliding dot products & statistics for each tile is overkill
        compute_statistics(T_A_dev[device], norms_A[device], df_A[device], dg_A[device], means_A[device], t_n_x, m, streams.at(device), scratchpad[device]);
        gpuErrchk(hipPeekAtLastError());
        compute_statistics(T_B_dev[device], norms_B[device], df_B[device], dg_B[device],  means_B[device], t_n_y, m, streams.at(device), scratchpad[device]);
        gpuErrchk(hipPeekAtLastError());
        thrust::device_ptr<unsigned long long int> ptr_A = thrust::device_pointer_cast(profile_A_merged[device]);
        thrust::device_ptr<unsigned long long int> ptr_B = thrust::device_pointer_cast(profile_B_merged[device]);
        thrust::transform(thrust::cuda::par.on(streams.at(device)), profile_A_dev[device], profile_A_dev[device] + t_n_x, profile_idx_A_dev[device], profile_A_merged[device], combiner);
        gpuErrchk(hipPeekAtLastError());
        thrust::transform(thrust::cuda::par.on(streams.at(device)), profile_B_dev[device], profile_B_dev[device] + t_n_y, profile_idx_B_dev[device], profile_B_merged[device], combiner);
        gpuErrchk(hipPeekAtLastError());

        SCRIMP_Tile tile(t, T_A_dev[device], T_B_dev[device], df_A[device], df_B[device], dg_A[device], dg_B[device], norms_A[device], norms_B[device], means_A[device], means_B[device],  QT_dev[device], profile_A_merged[device], profile_B_merged[device], start_x, start_y, t_size_y, t_size_x, m, scratch[device]);
        hipEventRecord(clocks_start[device], streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        err = tile.execute(streams.at(device));
        hipEventRecord(clocks_end[device], streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        return err; 

}

void SCRIMP_Operation::get_tile_ordering(list<pair<int,int>> &tile_ordering) {
	size_t num_tile_rows = ceil((size_A - m + 1) / (float) tile_n);
	size_t num_tile_cols = ceil((size_B - m + 1) / (float) tile_n);

	for(int offset = 0; offset < num_tile_rows - 1; ++offset) {
		for(int diag = 0; diag < num_tile_cols - 1 - offset; ++diag) {
			tile_ordering.emplace_back(diag,diag + offset);
		}
	}

	for(int i = 0; i < num_tile_rows; ++i) {
		tile_ordering.emplace_back(i, num_tile_cols - 1);
	}


}


bool SCRIMP_Operation::pick_and_start_next_tile_self_join(int dev, list<pair<int,int>> &tile_order, const vector<double> &T_h, const vector<float> &profile_h, const vector<unsigned int> &profile_idx_h, size_t &size_x, size_t &size_y, size_t &start_x, size_t &start_y)
{
    
    bool done = false;
    int tile_row = tile_order.front().first;
    int tile_col = tile_order.front().second;
    start_x = tile_col * tile_n;
    start_y = tile_row * tile_n;
    size_x = min(tile_size, size_A - start_x);
    size_y = min(tile_size, size_B - start_y);
    if(tile_row == tile_col) {
        //partial tile on diagonal
        do_tile(SELF_JOIN_UPPER_TRIANGULAR, size_x, size_y, start_x, start_y, dev, T_h, profile_h, profile_idx_h);
    } else {
        // full tile
        do_tile(SELF_JOIN_FULL_TILE, size_x, size_y, start_x, start_y, dev, T_h, profile_h, profile_idx_h);
    }
    tile_order.pop_front();
    if(tile_order.empty()){
        done = true;
    }
    return done;
}


void merge_partial_on_host(vector<unsigned long long int> &profile_to_merge, vector<float> &merge_target, vector<unsigned int> &merge_idx_target, size_t merge_start, size_t tile_sz)
{
    auto iter_begin = thrust::make_zip_iterator(thrust::make_tuple(merge_target.data() + merge_start, merge_idx_target.data() + merge_start, profile_to_merge.data()));
    auto iter_end = thrust::make_zip_iterator(thrust::make_tuple(merge_target.data() + merge_start + tile_sz, merge_idx_target.data() + merge_start + tile_sz, profile_to_merge.data() + tile_sz));
    thrust::for_each(iter_begin, iter_end, max_with_index());

}

SCRIMPError_t SCRIMP_Operation::do_self_join(const vector<double> &T_host, vector<float> &profile, vector<unsigned int> &profile_idx)
{
    list<pair<int,int>> tile_ordering;
    vector< vector<unsigned long long int> > profileA_h(devices.size(), vector<unsigned long long int>(tile_n)), profileB_h(devices.size(), vector<unsigned long long int>(tile_n));
    bool done = false;
    int last_dev;
    vector<size_t> n_x(devices.size());
    vector<size_t> n_y(devices.size());
    vector<size_t> n_x_2(devices.size());
    vector<size_t> n_y_2(devices.size());
    vector<size_t> pos_x(devices.size());
    vector<size_t> pos_y(devices.size());
    vector<size_t> pos_x_2(devices.size());
    vector<size_t> pos_y_2(devices.size());
    
    get_tile_ordering(tile_ordering);
    printf("Performing self join with %lu tiles.\n", tile_ordering.size() );
    size_t total_tiles = tile_ordering.size();
    size_t completed_tiles = 0;
    for(auto device : devices) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        done = pick_and_start_next_tile_self_join(device, tile_ordering, T_host, profile, profile_idx, n_x[device], n_y[device], pos_x[device], pos_y[device]);
        gpuErrchk(hipPeekAtLastError());
        if (done) {
            last_dev = device;
            break;
        }
    }

    while(!done) {
        for(auto device : devices) {
            hipSetDevice(device);
            gpuErrchk(hipPeekAtLastError());
            hipMemcpyAsync(profileA_h.at(device).data(), profile_A_merged[device], sizeof(unsigned long long int) * (n_x[device] - m + 1), hipMemcpyDeviceToHost, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
            hipMemcpyAsync(profileB_h.at(device).data(), profile_B_merged[device], sizeof(unsigned long long int) * (n_y[device] - m + 1), hipMemcpyDeviceToHost, streams.at(device));
            gpuErrchk(hipPeekAtLastError());
            hipEventRecord(copy_to_host_done[device], streams.at(device));
            gpuErrchk(hipPeekAtLastError());
            n_x_2[device] = n_x[device];
            n_y_2[device] = n_y[device];
            pos_x_2[device] = pos_x[device];
            pos_y_2[device] = pos_y[device];
            if(!done) {
                done = pick_and_start_next_tile_self_join(device, tile_ordering, T_host, profile, profile_idx, n_x[device], n_y[device], pos_x[device], pos_y[device]);
                if(done) {
                    last_dev = device;
                }
            }
        }

        for(auto device : devices) {
            hipSetDevice(device);
            gpuErrchk(hipPeekAtLastError());
            hipEventSynchronize(copy_to_host_done[device]);
            gpuErrchk(hipPeekAtLastError());
            merge_partial_on_host(profileA_h.at(device), profile, profile_idx, pos_x_2[device], (n_x_2[device] - m + 1));
            gpuErrchk(hipPeekAtLastError());
            merge_partial_on_host(profileB_h.at(device), profile, profile_idx, pos_y_2[device], (n_y_2[device] - m + 1));
            gpuErrchk(hipPeekAtLastError());
            completed_tiles++;
            printf("%f percent complete\n", (completed_tiles / (float) total_tiles) * 100);
            
        }


    }

    for(int device = 0; device <= last_dev; ++device) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profileA_h.at(device).data(), profile_A_merged[device], sizeof(unsigned long long int) * (n_x[device] - m + 1), hipMemcpyDeviceToHost, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipMemcpyAsync(profileB_h.at(device).data(), profile_B_merged[device], sizeof(unsigned long long int) * (n_y[device] - m + 1), hipMemcpyDeviceToHost, streams.at(device));
        gpuErrchk(hipPeekAtLastError());
        hipEventRecord(copy_to_host_done[device], streams.at(device));
        gpuErrchk(hipPeekAtLastError());
    }
    for(int device = 0; device <= last_dev; ++device) {
        hipSetDevice(device);
        gpuErrchk(hipPeekAtLastError());
        hipEventSynchronize(copy_to_host_done[device]);
        gpuErrchk(hipPeekAtLastError());
        merge_partial_on_host(profileA_h.at(device), profile, profile_idx, pos_x[device], (n_x[device] - m + 1));
        gpuErrchk(hipPeekAtLastError());
        merge_partial_on_host(profileB_h.at(device), profile, profile_idx, pos_y[device], (n_y[device] - m + 1));
        gpuErrchk(hipPeekAtLastError());
        completed_tiles++;
        printf("%f percent complete\n", (completed_tiles / (float) total_tiles) * 100);
    }
    return SCRIMP_NO_ERROR;
}

void do_SCRIMP(const vector<double> &T_h, vector<float> &profile_h, vector<unsigned int> &profile_idx_h, const unsigned int m, const size_t max_tile_size, const vector<int> &devices) {
    if(devices.empty()) {
        printf("Error: no gpu provided\n");
        exit(0);
    }
    // Allocate and initialize memory
    clock_t start, end;
    SCRIMP_Operation op(T_h.size(), T_h.size(), m, max_tile_size, devices);
    op.init();
    gpuErrchk(hipPeekAtLastError());
    start = clock();
    op.do_self_join(T_h, profile_h, profile_idx_h);
    hipDeviceSynchronize();
    end = clock();
    gpuErrchk(hipPeekAtLastError());
    op.destroy();
    gpuErrchk(hipPeekAtLastError());
    printf("Finished SCRIMP to generate partial matrix profile of size %lu in %f seconds on %lu devices:\n", profile_h.size(), (end - start) / (double) CLOCKS_PER_SEC, devices.size());
}

//Reads input time series from file
template<class DTYPE>
void readFile(const char* filename, vector<DTYPE>& v, const char *format_str) 
{
    FILE* f = fopen( filename, "r");
    if(f == NULL){
        printf("Unable to open %s for reading, please make sure it exists\n", filename);
        exit(0);
    }
    DTYPE num;
    while(!feof(f)){
            fscanf(f, format_str, &num);
            v.push_back(num);
        }
    v.pop_back();
    fclose(f);
}
    
}

int main(int argc, char** argv) {

    if(argc < 5) {
        printf("Usage: SCRIMP <window_len> <max_tile_size> <input file> <profile output file> <index output file> [Optional: list of GPU device numbers to run on]\n");
        exit(0);
    }

    int window_size = atoi(argv[1]);
    int max_tile_size = atoi(argv[2]);
    vector<double> T_h;
    SCRIMP::readFile<double>(argv[3], T_h, "%lf");
    int n = T_h.size() - window_size + 1;
    vector<float> profile(n, CC_MIN);
    vector<unsigned int> profile_idx(n, 0);
     
    hipFree(0);
    
    vector<int> devices;
    
    if(argc == 6) {
        // Use all available devices 
        int num_dev;
        hipGetDeviceCount(&num_dev);
        for(int i = 0; i < num_dev; ++i){ 
            devices.push_back(i);
        }
    } else {
        // Use the devices specified
        int x = 6;
        while (x < argc) {
            devices.push_back(atoi(argv[x]));
            ++x;
        }
    }
    
    printf("Starting SCRIMP\n");
     
    SCRIMP::do_SCRIMP(T_h, profile, profile_idx, window_size, max_tile_size, devices);
    
    printf("Now writing result to files\n");
    FILE* f1 = fopen( argv[4], "w");
    FILE* f2 = fopen( argv[5], "w");
    for(int i = 0; i < profile.size(); ++i){
         fprintf(f1, "%f\n", sqrt(max(2*window_size*(1 - profile[i]), 0.0)));
         fprintf(f2, "%u\n", profile_idx[i] + 1);
    }
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipDeviceReset());
    fclose(f1);
    fclose(f2);
    printf("Done\n");
    return 0;
}

