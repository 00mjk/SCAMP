#include "hip/hip_runtime.h"
#include "fft_helper.h"
#include <hipfft/hipfft.h> 

namespace SCRIMP {

#define WORK_SIZE 512

template<class DTYPE>
__global__ void elementwise_multiply_inplace(const DTYPE* A, DTYPE *B, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size) {
       B[tid] *= A[tid];
    }
} 

template<>
__global__ void elementwise_multiply_inplace(const hipDoubleComplex* A, hipDoubleComplex* B, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size) {
       B[tid] = hipCmul(A[tid], B[tid]);
    }
}

template<>
__global__ void elementwise_multiply_inplace(const hipComplex* A, hipComplex* B, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size) {
       B[tid] = hipCmulf(A[tid], B[tid]);
    }
}

// A is input unaligned sliding dot products produced by ifft
// out is the computed vector of distances
template<class DTYPE>
__global__ void normalized_aligned_dot_products(const DTYPE* A, const DTYPE divisor,
                                                const unsigned int m, const unsigned int n,
                                                DTYPE* QT)
{
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    if (a < n) {
        QT[a] = A[a + m - 1] / divisor;
    }
}

template<class DTYPE>
__global__ void populate_reverse_pad(const DTYPE *Q, DTYPE *Q_reverse_pad, const int window_size, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < window_size) {
        Q_reverse_pad[tid] = Q[window_size - 1 - tid];
    }else if(tid < size){ 
        Q_reverse_pad[tid] = 0;
    }
}

template<class DATATYPE, class CUFFT_DTYPE>
SCRIMPError_t fft_precompute_helper<DATATYPE, CUFFT_DTYPE>::compute_QT(DATATYPE* QT, const DATATYPE* T, const DATATYPE *Q, hipStream_t s)
{        

    hipfftResult cufftError;
    hipError_t error;

    const int n = size - window_size + 1;
    dim3 block(WORK_SIZE, 1, 1);
    

    cufftError = hipfftSetStream(fft_plan, s);
    if (cufftError != HIPFFT_SUCCESS) {
        return SCRIMP_CUFFT_ERROR;
    }
    cufftError = hipfftSetStream(ifft_plan,s);
    if (cufftError != HIPFFT_SUCCESS) {
        return SCRIMP_CUFFT_ERROR;
    }
    
    // Compute the FFT of the time series
    if (double_precision) {
        cufftError = hipfftExecD2Z(fft_plan, const_cast<DATATYPE*>(T), Tc);
    } else {
        cufftError = hipfftExecR2C(fft_plan, const_cast<DATATYPE*>(T), Tc);
    }

    if (cufftError != HIPFFT_SUCCESS) {
        return SCRIMP_CUFFT_EXEC_ERROR;
    }
    
    // Reverse and zero pad the query
    populate_reverse_pad<DATATYPE><<<dim3(ceil(size / (float) WORK_SIZE),1,1), block, 0, s>>>(Q, Q_reverse_pad, window_size, size);
    error = hipPeekAtLastError();
    if (error != hipSuccess) {
        return SCRIMP_CUDA_ERROR;
    }
    
    // Compute the FFT of the query
    if (double_precision) {
        cufftError = hipfftExecD2Z(fft_plan, Q_reverse_pad, Qc);
    } else {
        cufftError = hipfftExecR2C(fft_plan, Q_reverse_pad, Qc);
    }
    if (cufftError != HIPFFT_SUCCESS) {
        return SCRIMP_CUFFT_EXEC_ERROR;
    }
    
    elementwise_multiply_inplace<CUFFT_DTYPE><<<dim3(ceil(cufft_data_size / (float) WORK_SIZE), 1, 1), block, 0, s>>>(Tc, Qc, cufft_data_size);
    error = hipPeekAtLastError();
    if ( error != hipSuccess) {
        return SCRIMP_CUDA_ERROR;
    }

    // Compute the ifft
    // Use the space for the query as scratch space as we no longer need it
    if (double_precision) {
        cufftError = hipfftExecZ2D(ifft_plan, Qc, Q_reverse_pad);
    } else {
        cufftError = hipfftExecC2R(ifft_plan, Qc, Q_reverse_pad);
    }

    if (cufftError != HIPFFT_SUCCESS) {
        return SCRIMP_CUFFT_EXEC_ERROR;
    }
    
    normalized_aligned_dot_products<DATATYPE><<<dim3(ceil(n / (float) WORK_SIZE), 1, 1), block, 0, s>>>(Q_reverse_pad, size, window_size, n, QT);
    error = hipPeekAtLastError();

    if(error != hipSuccess) {
        return SCRIMP_CUDA_ERROR;
    }

    return SCRIMP_NO_ERROR;
    
}

} 
